
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__ void mykernel(int *data){

  (*data)++;
}

int main(void)
{
    int numDevices;
    if (hipGetDeviceCount(&numDevices) != hipSuccess) {
        fprintf(stderr, "Error calling cudaGetDeviceCount\n");
        return -1;
    }
    printf("found %d devices\n", numDevices);
    hipSetDevice(0);
    int *d_data, h_data = 0;
    hipMalloc((void **)&d_data, sizeof(int));
    hipMemcpy(d_data, &h_data, sizeof(int), hipMemcpyHostToDevice);
    mykernel<<<1,1>>>(d_data);
    hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost);
    printf("data = %d\n", h_data);
    printf("Exceuted\n");
    return 0;
}
